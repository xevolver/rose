#include "test2016_01.h"
#include <stdio.h>

#define TEST_VEC_LEN 10

/* module load cudatoolkit/7.5 */
/* use gcc-4.9.3p */
/* nvcc -O2 --expt-extended-lambda -arch compute_35 -std=c++11 main.cu */

int foobar(int i);

int main(int argc, char *argv[])
{
   int *value ;

   hipMallocManaged((void **)&value, sizeof(int)*TEST_VEC_LEN, hipMemAttachGlobal) ;

// C++11 CUDA code works without the "__device__" keyword in front of the lambda.
// forall(cuda_traversal(), TEST_VEC_LEN, [=] __device__ (int i) { value[i] = i ; } ) ;
// forall(cuda_traversal(), TEST_VEC_LEN, [=] (int i) { value[i] = i ; } ) ;

// forall(cuda_traversal(), TEST_VEC_LEN, [=] __attribute__(my_device) (int i) { value[i] = i ; } ) ;
// forall(cuda_traversal(), TEST_VEC_LEN, [=] (int i) { value[i] = i ; } ) ;
// forall(cuda_traversal(), TEST_VEC_LEN, [=] __attribute__((my_device)) (int i) { value[i] = i ; } ) ;
// forall(cuda_traversal(), TEST_VEC_LEN, __attribute__((my_device)) [=] (int i) { value[i] = i ; } ) ;
   forall(cuda_traversal(), TEST_VEC_LEN, [=] (int i) __attribute__((my_device)) { value[i] = i ; } ) ;

   int x __attribute__((my_device));

   hipDeviceSynchronize() ;

   for (int i=0; i<TEST_VEC_LEN; ++i) {
      printf("%d ", value[i]) ;
   }
   printf("\n") ;

   return 0 ;
}

