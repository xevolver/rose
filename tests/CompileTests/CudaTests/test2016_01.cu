#include "test2016_01.h"
// #include <stdio.h>

#define TEST_VEC_LEN 10

/* module load cudatoolkit/7.5 */
/* use gcc-4.9.3p */
/* nvcc -O2 --expt-extended-lambda -arch compute_35 -std=c++11 main.cu */

int foobar(int i);

int main(int argc, char *argv[])
{
   int *value ;

#if 1
   hipMallocManaged((void **)&value, sizeof(int)*TEST_VEC_LEN, hipMemAttachGlobal) ;
#endif
#if 0
// This is what we want to generate in ROSE.
// This code is a problem for ROSE error on "[=]" syntax.
   forall(cuda_traversal(), TEST_VEC_LEN, [=] __device__ (int i) {
       value[i] = i ;
   } ) ;
#else
// This is compilable by ROSE (but not relevant).
   forall(cuda_traversal(), TEST_VEC_LEN, [=] (int i) __attribute__((my_device)) {
       value[i] = i ;
   } ) ;
#endif

   hipDeviceSynchronize() ;

#if 0
   for (int i=0; i<TEST_VEC_LEN; ++i) {
      printf("%d ", value[i]) ;
   }
   printf("\n") ;
#endif

   return 0 ;
}

