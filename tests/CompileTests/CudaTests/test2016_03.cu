#include "test2016_03.h"
#include <stdio.h>

#define TEST_VEC_LEN 10

/* module load cudatoolkit/7.5 */
/* use gcc-4.9.3p */
/* nvcc -O2 --expt-extended-lambda -arch compute_35 -std=c++11 main.cu */

int main(int argc, char *argv[])
{
   int *value ;

   hipMallocManaged((void **)&value,
                     sizeof(int)*TEST_VEC_LEN,
                     hipMemAttachGlobal) ;

// This is what we want users to write. In the generated rose_test2016_03.cu 
// file this will have the added "__device__" keyword added.
   forall(cuda_traversal(), TEST_VEC_LEN, [=] (int i) {
       value[i] = i ;
   } ) ;

   hipDeviceSynchronize() ;

   for (int i=0; i<TEST_VEC_LEN; ++i) {
      printf("%d ", value[i]) ;
   }
   printf("\n") ;

   return 0 ;
}

